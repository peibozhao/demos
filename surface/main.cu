#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime_api.h>
#include <surface_functions.h>

#define CHECK(cu_ret) \
  if (cu_ret != hipSuccess) { \
    std::cout << "cuda error: " << cu_ret << std::endl; \
    exit(-1); \
  }

// Simple transformation kernel
__global__ void transformKernel(float* output,
                                hipSurfaceObject_t surfObj,
                                int width, int height)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    surf2Dread(output + y * width + x, surfObj, x * 4 + 4, y, hipBoundaryModeClamp);
}

// Host code
int main()
{
    const int height = 16;
    const int width = 16;

    float *h_data = (float *)std::malloc(sizeof(float) * width * height);
    for (int i = 0; i < height; ++i) {
      for (int j = 0; j < width; ++j) {
        h_data[i * width + j] = i + j;
        std::cout << h_data[i * width + j] << "\t";
      }
      std::cout << std::endl;
    }

    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    CHECK(hipMallocArray(&cuArray, &channelDesc, width, height, hipArraySurfaceLoadStore));

    const size_t spitch = width * sizeof(float);
    CHECK(hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float),
                        height, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    hipSurfaceObject_t surfObj;
    CHECK(hipCreateSurfaceObject(&surfObj, &resDesc));

    float *output;
    CHECK(hipMalloc(&output, width * height * sizeof(float)));

    // Invoke kernel
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                    (height + threadsperBlock.y - 1) / threadsperBlock.y);
    transformKernel<<<numBlocks, threadsperBlock>>>(output, surfObj, width, height);

    float *h_ret = (float *)std::malloc(sizeof(float) * width * height);
    memset(h_ret, 0, sizeof(float) * width * height);
    // Copy data from device back to host
    CHECK(hipMemcpy(h_ret, output, width * height * sizeof(float),
                hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());
    std::cout << "----" << std::endl;
    for (int i = 0; i < height; ++i) {
      for (int j = 0; j < width; ++j) {
        std::cout << h_ret[i * width + j] << "\t";
      }
      std::cout << std::endl;
    }


    CHECK(hipDestroySurfaceObject(surfObj));

    // Free device memory
    CHECK(hipFreeArray(cuArray));
    CHECK(hipFree(output));

    // Free host memory
    free(h_data);

    return 0;
}
