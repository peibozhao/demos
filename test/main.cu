#include "hip/hip_runtime.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <vector>
#include <algorithm>
#include <numeric>
#include <fstream>

__global__ void kern(float *arr, float *arr2, int size, float *ret) {
  int x = blockIdx.x * gridDim.x + threadIdx.x;
  if (x < size) {
    ret[x] = arr[x] * arr2[x];
  }
}

int main(int argc, char *argv[]) {

  std::vector<float> vv;
  std::ifstream ifs("test.txt");
  while (ifs) {
    float f;
    ifs >> f;
    if (!ifs) break;
    vv.emplace_back(f);
    std::cout << vv.back() << std::endl;
  }
  return 0;


  std::vector<float> v(100, 0.1);
  std::vector<float> v2(100, 9371);

  std::iota(v.begin(), v.end(), 10);
  std::iota(v2.begin(), v2.end(), 7);

  thrust::device_vector<float> dv(v);
  thrust::device_vector<float> dv2(v2);
  thrust::device_vector<float> dr(100);

  hipDeviceSynchronize();
  kern<<<512, 512>>>(dv.data().get(), dv2.data().get(), 100, dr.data().get());
  hipDeviceSynchronize();

  for (int i = 0; i < 100; ++i) {
    std::cout << dr[i] << std::endl;
  }

  return 0;
}
