#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>
#include <iostream>
#include <surface_functions.h>

#define CHECK(cu_ret) \
  if (cu_ret != hipSuccess) { \
    std::cout << "line: " << __LINE__ << ". cuda error: " << cu_ret << std::endl; \
    exit(-1); \
  }

// Simple transformation kernel
__global__ void transformKernel(float* output,
                                hipTextureObject_t texObj,
                                int width, int height,
                                float delta)
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = (x + delta) / (float)width;
    float v = (y + delta) / (float)height;

    // printf("%f %f\n", u, v);

    // Read from texture and write to global memory
    output[y * width + x] = tex2D<float>(texObj, u, v);
}

// Simple transformation kernel
__global__ void transformKernel2(float* output,
                                hipSurfaceObject_t surObj,
                                int width, int height,
                                float delta)
{
    // Calculate normalized texture coordinates
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // float u = (x + delta) / (float)width;
    // float v = (y + delta) / (float)height;

    // printf("%f %f\n", u, v);

    // Read from texture and write to global memory
    // output[y * width + x] = tex2D<float>(texObj, u, v);
    // surf2Dread<float>(output + y * width + x, texObj, u, v);
    surf2Dread(&output[y * width * x], surObj, x * 4, y, hipBoundaryModeTrap);
}

// Host code
int main()
{
    // const int height = 1024;
    // const int width = 1024;
    const int height = 16;
    const int width = 16;
    float delta = 1.0;

    // Allocate and set some host data
    float *h_data = (float *)std::malloc(sizeof(float) * width * height);
    for (int i = 0; i < height; ++i) {
      for (int j = 0; j < width; ++j) {
        h_data[i * width + j] = i + j;
        std::cout << h_data[i * width + j] << "\t";
      }
      std::cout << std::endl;
    }

    // Allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    CHECK(hipMallocArray(&cuArray, &channelDesc, width, height, hipArraySurfaceLoadStore));

    // Set pitch of the source (the width in memory in bytes of the 2D array pointed
    // to by src, including padding), we dont have any padding
    // pitch = stride = lda
    const size_t spitch = width * sizeof(float);
    // Copy data located at address h_data in host memory to device memory
    CHECK(hipMemcpy2DToArray(cuArray, 0, 0, h_data, spitch, width * sizeof(float),
                        height, hipMemcpyHostToDevice));

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    // texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // Create texture object
    // hipTextureObject_t texObj = 0;
    // CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));
    hipSurfaceObject_t surObj = 0;
    CHECK(hipCreateSurfaceObject(&surObj, &resDesc));

    // Allocate result of transformation in device memory
    float *output;
    CHECK(hipMalloc(&output, width * height * sizeof(float)));

    // Invoke kernel
    dim3 threadsperBlock(16, 16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                    (height + threadsperBlock.y - 1) / threadsperBlock.y);
    transformKernel2<<<numBlocks, threadsperBlock>>>(output, surObj, width, height, delta);

    float *h_ret = (float *)std::malloc(sizeof(float) * width * height);
    memset(h_ret, 0, sizeof(float) * width * height);
    // Copy data from device back to host
    CHECK(hipMemcpy(h_ret, output, width * height * sizeof(float),
                hipMemcpyDeviceToHost));

    CHECK(hipDeviceSynchronize());
    std::cout << "----" << std::endl;
    for (int i = 0; i < height; ++i) {
      for (int j = 0; j < width; ++j) {
        std::cout << h_ret[i * width + j] << "\t";
      }
      std::cout << std::endl;
    }


    // Destroy texture object
    // CHECK(hipDestroyTextureObject(texObj));

    // Free device memory
    CHECK(hipFreeArray(cuArray));
    CHECK(hipFree(output));

    // Free host memory
    free(h_data);

    return 0;
}
