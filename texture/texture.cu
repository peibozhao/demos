
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

constexpr int height = 1536;
constexpr int width = 1920;

__global__ void undistort2(const hipTextureObject_t dev_src, const float *dev_map1, const float *dev_map2, uint8_t *dev_dst) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float map_x = dev_map1[idx];
  float map_y = dev_map2[idx];

  // int x1 = int(map_x);
  // int x2 = x1 + 1;
  // int y1 = int(map_y);
  // int y2 = y1 + 1;

  // float scale[4] = {(x2 - map_x) * (y2 - map_y), (map_x - x1) * (y2 - map_y),
  //   (x2 - map_x) * (map_y - y1), (map_x - x1) * (map_y - y1)};

  // // uchar4 gather = tex2Dgather<uchar4>(dev_src, map_x, map_y);
  // // dev_dst[idx] = scale[0] * gather.x + scale[1] * gather.y + scale[2] * gather.z + scale[3] * gather.w;

  // uint8_t p[4] = {tex2D<uint8_t>(dev_src, x1, y1), tex2D<uint8_t>(dev_src, x1, y2), tex2D<uint8_t>(dev_src, x2, y1), tex2D<uint8_t>(dev_src, x2, y2)};

  // dev_dst[idx] = scale[0] * p[0] + scale[1] * p[1] + scale[2] * p[2] + scale[3] * p[3];

  dev_dst[idx] = tex2D<float>(dev_src, map_x, map_y);
}

int main(int argc, char *argv[])
{
  std::ifstream map1_fstream("map1.txt");
  std::ifstream map2_fstream("map2.txt");
  float *map1 = new float[height * width];
  float *map2 = new float[height * width];
  for (int i = 0; i < height * width; ++i) {
    // map1_fstream >> map1[i];
    // map2_fstream >> map2[i];

    float f;
    map1_fstream >> f;
    map1[i] = f;
    map2_fstream >> f;
    map2[i] = f;

    // std::cout << map1[i] << std::endl;
  }

  uint8_t *src = new uint8_t[height * width];
  std::ifstream src_fstream("test.gray", std::ios::binary);
  if (!src_fstream.is_open()) {
    std::cout << "file is not open" << std::endl;
    return -1;
  }
  src_fstream.read((char *)src, height * width);
  src_fstream.close();

  float *src_f = new float[height * width];
  for (int i = 0; i < height * width; ++i) {
    src_f[i] = src[i];
  }

  // cudaChannelFormatDesc desc = cudaCreateChannelDesc<uint8_t>(width, height, 0, 0, cudaChannelFormatKindUnsigned);
  hipChannelFormatDesc desc = hipCreateChannelDesc<uint8_t>();
  // cudaChannelFormatDesc desc = cudaCreateChannelDesc<float>();

  hipError_t err;
  hipArray_t dev_src;
  err = hipMallocArray(&dev_src, &desc, width, height);
  // err = cudaMallocArray(&dev_src, &desc, width, height, cudaArrayTextureGather);
  std::cout << "hipMallocArray " << err << std::endl;

  err = hipMemcpy2DToArray(dev_src, 0, 0, src, width, width, height, hipMemcpyHostToDevice);
  // err = cudaMemcpy2DToArray(dev_src, 0, 0, src_f, width * sizeof(float), width * sizeof(float), height, cudaMemcpyHostToDevice);
  std::cout << "hipMemcpy2DToArray " << err << std::endl;

  hipTextureObject_t texture_src;
  hipResourceDesc resource_desc;
  resource_desc.resType = hipResourceTypeArray;
  resource_desc.res.array.array = dev_src;
  hipTextureDesc texture_desc;
  memset(&texture_desc, 0, sizeof(texture_desc));
  texture_desc.addressMode[0] = hipTextureAddressMode::hipAddressModeClamp;
  texture_desc.addressMode[1] = hipTextureAddressMode::hipAddressModeClamp;
  texture_desc.filterMode = hipTextureFilterMode::hipFilterModePoint;
  texture_desc.readMode = hipTextureReadMode::hipReadModeElementType;
  texture_desc.normalizedCoords = 0;


  err = hipCreateTextureObject(&texture_src, &resource_desc, &texture_desc, nullptr);

  std::cout << "hipCreateTextureObject " << err << std::endl;
  // char *dev_src;
  // cudaMalloc(&dev_src, height * width);
  // cudaMemcpy(dev_src, src, height * width, cudaMemcpyHostToDevice);

  float *dev_map1, *dev_map2;
  hipMalloc(&dev_map1, height * width * sizeof(float));
  hipMalloc(&dev_map2, height * width * sizeof(float));

  hipMemcpy(dev_map1, map1, height * width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_map2, map2, height * width * sizeof(float), hipMemcpyHostToDevice);


  uint8_t *dst = new uint8_t[height * width];
  uint8_t *dev_dst;
  hipMalloc(&dev_dst, height * width * sizeof(uint8_t));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i = 0; i < 10; ++i) {
    hipDeviceSynchronize();
    auto s = std::chrono::system_clock::now();
    hipEventRecord(start);
    undistort2<<<1536 * 1920 / 512, 512>>>(texture_src, dev_map1, dev_map2, dev_dst);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    auto e = std::chrono::system_clock::now();
    hipError_t err = hipGetLastError();
    std::cout << "Cost: " << std::chrono::duration_cast<std::chrono::microseconds>(e - s).count() << "us " << err << std::endl;

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << ms << std::endl;
  }

  hipMemcpy(dst, dev_dst, height * width, hipMemcpyDeviceToHost);


  std::ofstream ofs("output.gray", std::ios::binary);
  ofs.write((char *)dst, width * height);
  ofs.close();

  return 0;
}
