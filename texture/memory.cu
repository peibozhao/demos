
#include <chrono>
#include <hip/hip_runtime_api.h>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>

constexpr int height = 1536;
constexpr int width = 1920;

__global__ void undistort(const float *dev_src, const float *dev_map1, const float *dev_map2, uint8_t *dev_dst) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  float map_x = dev_map1[idx];
  float map_y = dev_map2[idx];

  int x1 = int(map_x);
  int x2 = x1 + 1;
  int y1 = int(map_y);
  int y2 = y1 + 1;

  float scale[4] = {(x2 - map_x) * (y2 - map_y), (map_x - x1) * (y2 - map_y),
    (x2 - map_x) * (map_y - y1), (map_x - x1) * (map_y - y1)};

  float p[4] = {dev_src[idx], dev_src[idx + 1], dev_src[idx + width], dev_src[idx + width + 1]};

  dev_dst[idx] = scale[0] * p[0] + scale[1] * p[1] + scale[2] * p[2] + scale[3] * p[3];

  // int map_x = dev_map1[idx];
  // int map_y = dev_map2[idx];

  // dev_dst[idx] = dev_src[map_y * width + map_x];
}

int main(int argc, char *argv[])
{
  std::ifstream map1_fstream("map1.txt");
  std::ifstream map2_fstream("map2.txt");
  float *map1 = new float[height * width];
  float *map2 = new float[height * width];
  for (int i = 0; i < height * width; ++i) {
    // map1_fstream >> map1[i];
    // map2_fstream >> map2[i];

    float f;
    map1_fstream >> f;
    map1[i] = f;
    map2_fstream >> f;
    map2[i] = f;

    // std::cout << map1[i] << std::endl;
  }

  uint8_t *src = new uint8_t[height * width];
  std::ifstream src_fstream("test.gray", std::ios::binary);
  if (!src_fstream.is_open()) {
    std::cout << "file is not open" << std::endl;
    return -1;
  }
  src_fstream.read((char *)src, height * width);
  src_fstream.close();

  float *src_f = new float[height * width];
  for (int i = 0; i < height * width; ++i) {
    src_f[i] = src[i];
  }

  // uint8_t *dev_src;
  // cudaMalloc(&dev_src, height * width);
  // cudaMemcpy(dev_src, src, height * width, cudaMemcpyHostToDevice);
  float *dev_src;
  hipMalloc(&dev_src, height * width * sizeof(float));
  hipMemcpy(dev_src, src_f, height * width * sizeof(float), hipMemcpyHostToDevice);

  float *dev_map1, *dev_map2;
  hipMalloc(&dev_map1, height * width * sizeof(float));
  hipMalloc(&dev_map2, height * width * sizeof(float));

  hipMemcpy(dev_map1, map1, height * width * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_map2, map2, height * width * sizeof(float), hipMemcpyHostToDevice);


  uint8_t *dst = new uint8_t[height * width];
  uint8_t *dev_dst;
  hipMalloc(&dev_dst, height * width * sizeof(uint8_t));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for (int i = 0; i < 10; ++i) {
    hipDeviceSynchronize();
    auto s = std::chrono::system_clock::now();
    hipEventRecord(start);
    undistort<<<1536 * 1920 / 512, 512>>>(dev_src, dev_map1, dev_map2, dev_dst);
    hipEventRecord(stop);
    hipDeviceSynchronize();
    auto e = std::chrono::system_clock::now();
    hipError_t err = hipGetLastError();
    std::cout << "Cost: " << std::chrono::duration_cast<std::chrono::microseconds>(e - s).count() << "us " << err << std::endl;

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << ms << std::endl;
  }

  hipMemcpy(dst, dev_dst, height * width, hipMemcpyDeviceToHost);

  std::ofstream ofs("output.gray", std::ios::binary);
  ofs.write((char *)dst, width * height);
  ofs.close();

  return 0;
}
